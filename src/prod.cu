// #include <iostream>
// #include <cuda_runtime.h>

// #define BLOCK_SIZE 16

// // CUDA kernel for matrix-vector multiplication
// __global__ void matVecMulKernel(float *A, float *x, float *y, int rows, int cols) {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     if (row < rows) {
//         float sum = 0.0f;
//         for (int col = 0; col < cols; ++col) {
//             sum += A[row * cols + col] * x[col];
//         }
//         y[row] = sum;
//     }
// }

// void matVecMul(float *h_A, float *h_x, float *h_y, int rows, int cols) {
//     float *d_A, *d_x, *d_y;

//     // Allocate memory on the device
//     cudaMalloc((void**)&d_A, rows * cols * sizeof(float));
//     cudaMalloc((void**)&d_x, cols * sizeof(float));
//     cudaMalloc((void**)&d_y, rows * sizeof(float));

//     // Copy data from host to device
//     cudaMemcpy(d_A, h_A, rows * cols * sizeof(float), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_x, h_x, cols * sizeof(float), cudaMemcpyHostToDevice);

//     // Define the grid and block dimensions
//     dim3 dimBlock(1, BLOCK_SIZE);
//     dim3 dimGrid(1, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

//     // Launch the kernel
//     matVecMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, rows, cols);

//     // Copy the result back to the host
//     cudaMemcpy(h_y, d_y, rows * sizeof(float), cudaMemcpyDeviceToHost);

//     // Clean up
//     cudaFree(d_A);
//     cudaFree(d_x);
//     cudaFree(d_y);
// }

// int main() {
//     int rows = 4;
//     int cols = 4;

//     // Host input matrices
//     float h_A[] = {
//         1,  0,  1,  0,
//         0,  1,  0,  1,
//         1,  0, -1,  0,
//         0,  1,  0, -1};
//     float h_x[] = {1, 0, 0, 0};
//     float h_y[4];

//     // Matrix-vector multiplication
//     matVecMul(h_A, h_x, h_y, rows, cols);

//     // Print the result
//     for (int i = 0; i < rows; ++i) {
//         std::cout << h_y[i] << " ";
//     }
//     std::cout << std::endl;

//     return 0;
// }

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define BLOCK_SIZE 16

// CUDA kernel for complex matrix-vector multiplication
__global__ void matVecMulKernel(hipFloatComplex *A, hipFloatComplex *x, hipFloatComplex *y, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rows) {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int col = 0; col < cols; ++col) {
            sum = hipCaddf(sum, hipCmulf(A[row * cols + col], x[col]));
        }
        y[row] = sum;
    }
}

void matVecMul(hipFloatComplex *h_A, hipFloatComplex *h_x, hipFloatComplex *h_y, int rows, int cols) {
    hipFloatComplex *d_A, *d_x, *d_y;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, rows * cols * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_x, cols * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_y, rows * sizeof(hipFloatComplex));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, rows * cols * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, cols * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 dimBlock(1, BLOCK_SIZE);
    dim3 dimGrid(1, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the kernel
    matVecMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, rows, cols);

    // Copy the result back to the host
    hipMemcpy(h_y, d_y, rows * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}

int main() {
    int rows = 4;
    int cols = 4;

    // Host input matrices
    hipFloatComplex h_A[] = {
        make_hipFloatComplex(1, 0),  make_hipFloatComplex(0, 0),  make_hipFloatComplex(1, 0),  make_hipFloatComplex(0, 0),
        make_hipFloatComplex(0, 0),  make_hipFloatComplex(1, 0),  make_hipFloatComplex(0, 0),  make_hipFloatComplex(1, 0),
        make_hipFloatComplex(1, 0),  make_hipFloatComplex(0, 0), make_hipFloatComplex(-1, 0),  make_hipFloatComplex(0, 0),
        make_hipFloatComplex(0, 0),  make_hipFloatComplex(1, 0),  make_hipFloatComplex(0, 0), make_hipFloatComplex(-1, 0)
    };
    hipFloatComplex h_x[] = {make_hipFloatComplex(1, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0)};
    hipFloatComplex h_y[4];

    // Matrix-vector multiplication
    matVecMul(h_A, h_x, h_y, rows, cols);

    // Print the result
    for (int i = 0; i < rows; ++i) {
        //std::cout << "(" << cuCrealf(h_y[i]) << ", " << cuCimagf(h_y[i]) << ") ";
        std::cout << hipCrealf(h_y[i]) << " + " << hipCimagf(h_y[i]) << "i ";
    }
    std::cout << std::endl;

    return 0;
}

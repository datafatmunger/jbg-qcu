#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void tensorProductKernel(float* d_A, float* d_B, float* d_C, int aRows, int aCols, int bRows, int bCols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int totalRows = aRows * bRows;
    int totalCols = aCols * bCols;

    if (i < totalRows && j < totalCols) {
        int rowA = i / bRows;
        int colA = j / bCols;
        int rowB = i % bRows;
        int colB = j % bCols;
        d_C[i * totalCols + j] = d_A[rowA * aCols + colA] * d_B[rowB * bCols + colB];
    }
}

void tensorProduct(float* h_A, float* h_B, float* h_C, int aRows, int aCols, int bRows, int bCols) {
    int aSize = aRows * aCols * sizeof(float);
    int bSize = bRows * bCols * sizeof(float);
    int cSize = aRows * bRows * aCols * bCols * sizeof(float);

    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc((void**)&d_A, aSize);
    hipMalloc((void**)&d_B, bSize);
    hipMalloc((void**)&d_C, cSize);

    hipMemcpy(d_A, h_A, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((aCols * bCols + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (aRows * bRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    tensorProductKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows, bCols);

    hipMemcpy(h_C, d_C, cSize, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    const int aRows = 2;
    const int aCols = 2;
    const int bRows = 2;
    const int bCols = 2;

    float h_A[aRows * aCols] = {1, 1, 1, -1};
    float h_B[bRows * bCols] = {1, 0, 0, 1};
    float h_C[aRows * bRows * aCols * bCols];

    tensorProduct(h_A, h_B, h_C, aRows, aCols, bRows, bCols);

    for (int i = 0; i < aRows * bRows; ++i) {
        for (int j = 0; j < aCols * bCols; ++j) {
            std::cout << h_C[i * aCols * bCols + j] << " ";
        }
        std::cout << "\n";
    }

    return 0;
}

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <iostream>

__global__ void tensorProductKernel(hipFloatComplex* d_A, hipFloatComplex* d_B, hipFloatComplex* d_C, int aRows, int aCols, int bRows, int bCols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int totalRows = aRows * bRows;
    int totalCols = aCols * bCols;

    if (i < totalRows && j < totalCols) {
        int rowA = i / bRows;
        int colA = j / bCols;
        int rowB = i % bRows;
        int colB = j % bCols;
        d_C[i * totalCols + j] = hipCmulf(d_A[rowA * aCols + colA], d_B[rowB * bCols + colB]);
    }
}

void tensorProduct(hipFloatComplex* h_A, hipFloatComplex* h_B, hipFloatComplex* h_C, int aRows, int aCols, int bRows, int bCols) {
    int aSize = aRows * aCols * sizeof(hipFloatComplex);
    int bSize = bRows * bCols * sizeof(hipFloatComplex);
    int cSize = aRows * bRows * aCols * bCols * sizeof(hipFloatComplex);

    hipFloatComplex* d_A;
    hipFloatComplex* d_B;
    hipFloatComplex* d_C;

    hipMalloc((void**)&d_A, aSize);
    hipMalloc((void**)&d_B, bSize);
    hipMalloc((void**)&d_C, cSize);

    hipMemcpy(d_A, h_A, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((aCols * bCols + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (aRows * bRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    tensorProductKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows, bCols);

    hipMemcpy(h_C, d_C, cSize, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    const int aRows = 2;
    const int aCols = 2;
    const int bRows = 2;
    const int bCols = 2;

    hipFloatComplex h_A[aRows * aCols] = {make_hipFloatComplex(1, 0), make_hipFloatComplex(1, 0), make_hipFloatComplex(1, 0), make_hipFloatComplex(-1, 0)};
    hipFloatComplex h_B[bRows * bCols] = {make_hipFloatComplex(1, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(1, 0)};
    hipFloatComplex h_C[aRows * bRows * aCols * bCols];

    tensorProduct(h_A, h_B, h_C, aRows, aCols, bRows, bCols);

    for (int i = 0; i < aRows * bRows; ++i) {
        for (int j = 0; j < aCols * bCols; ++j) {
            std::cout << hipCrealf(h_C[i * aCols * bCols + j]) << "+" << hipCimagf(h_C[i * aCols * bCols + j]) << "i ";
        }
        std::cout << "\n";
    }

    return 0;
}

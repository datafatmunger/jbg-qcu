#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_complex.h>

#include <cmath>
#include <iostream>
#include <time.h>

// Error checking macro
#define cudaCheckError() {                                           \
    hipError_t e=hipGetLastError();                                \
    if(e!=hipSuccess) {                                             \
        std::cerr << "Cuda failure " << __FILE__ << ":" << __LINE__; \
        std::cerr << " '" << hipGetErrorString(e) << "'\n";         \
        exit(EXIT_FAILURE);                                          \
    }                                                                \
}

// Kernel to normalize the statevector
__global__ void normalize(hipFloatComplex* statevector, int len) {
    __shared__ float norm;
    if (threadIdx.x == 0) norm = 0.0f;
    __syncthreads();

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < len) {
        atomicAdd(&norm, hipCabsf(statevector[idx]) * hipCabsf(statevector[idx]));
    }
    __syncthreads();

    if (threadIdx.x == 0) norm = sqrtf(norm);
    __syncthreads();

    if (idx < len) {
        statevector[idx] = make_hipFloatComplex(hipCrealf(statevector[idx]) / norm, hipCimagf(statevector[idx]) / norm);
    }
}

// Kernel to compute the probabilities
__global__ void compute_probabilities(float* probabilities, hipFloatComplex* statevector, int len) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < len) {
        probabilities[idx] = hipCabsf(statevector[idx]) * hipCabsf(statevector[idx]);
    }
}

// Kernel to measure the statevector
__global__ void measure(int* result, float* probabilities, int len, unsigned long long seed) {
    hiprandState state;
    hiprand_init(seed, 0, 0, &state);
    float random_number = hiprand_uniform(&state);

    float cumulative_probability = 0.0f;
    for (int i = 0; i < len; ++i) {
        cumulative_probability += probabilities[i];
        if (random_number < cumulative_probability) {
            *result = i;
            break;
        }
    }
}

void checkNormalization(hipFloatComplex* statevector, int len) {
    float norm = 0.0;
    for (int i = 0; i < len; ++i) {
        norm += hipCabsf(statevector[i]) * hipCabsf(statevector[i]);
    }
    if (fabs(norm - 1.0) > 1e-6) {
        std::cerr << "Statevector must be normalized\n";
        exit(EXIT_FAILURE);
    }
}

int binaryStringToInt(const std::string& binaryStr) {
    int result = 0;
    for (char bit : binaryStr) {
        result <<= 1; // shift left by 1 bit
        result += (bit - '0'); // add the current bit
    }
    return result;
}

int test_measure(int shot) {
    int num_qubits = 2;
    int len = 1 << num_qubits;

    hipFloatComplex h_statevector[] = {
        make_hipFloatComplex(0.2, 0.0), make_hipFloatComplex(0.2, 0.0),
        make_hipFloatComplex(0.6, 0.0), make_hipFloatComplex(0.2, 0.0)
    };

    // // Calculate the norm of the statevector
    // float norm = 0.0f;
    // for (int i = 0; i < len; ++i) {
    //     float real_part = cuCrealf(h_statevector[i]);
    //     float imag_part = cuCimagf(h_statevector[i]);
    //     norm += real_part * real_part + imag_part * imag_part;
    // }
    // norm = sqrtf(norm);

    // // Normalize the statevector
    // for (int i = 0; i < len; ++i) {
    //     float real_part = cuCrealf(h_statevector[i]);
    //     float imag_part = cuCimagf(h_statevector[i]);
    //     h_statevector[i] = make_cuFloatComplex(real_part / norm, imag_part / norm);
    // }

    // // Verify normalization
    // float sum = 0.0f;
    // for (int i = 0; i < len; ++i) {
    //     float mag = cuCabsf(h_statevector[i]);
    //     sum += mag * mag;
    // }

    // Device memory allocations
    hipFloatComplex* d_statevector;
    float* d_probabilities;
    int* d_result;
    // Allocate memory for random number generator states
    //curandState *d_states;
    
    hipMalloc(&d_statevector, len * sizeof(hipFloatComplex));
    hipMalloc(&d_probabilities, len * sizeof(float));
    hipMalloc(&d_result, sizeof(int));
    //cudaMalloc((void **)&d_states, len * sizeof(curandState));

    // Copy statevector to device
    hipMemcpy(d_statevector, h_statevector, len * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Kernel launches
    int blockSize = 256;
    int gridSize = (len + blockSize - 1) / blockSize;

    // Generate a random seed
    unsigned long long seed = time(NULL) + shot;

    normalize<<<gridSize, blockSize>>>(d_statevector, len);
    cudaCheckError();

    compute_probabilities<<<gridSize, blockSize>>>(d_probabilities, d_statevector, len);
    cudaCheckError();

    measure<<<1, 1>>>(d_result, d_probabilities, len, seed);
    cudaCheckError();

    // Copy result back to host
    int h_result;
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Convert result to bitstring
    std::string bitstring;
    for (int i = num_qubits - 1; i >= 0; --i) {
        bitstring += ((h_result >> i) & 1) ? '1' : '0';
    }

    // Clean up
    hipFree(d_statevector);
    hipFree(d_probabilities);
    hipFree(d_result);

    return binaryStringToInt(bitstring);
}


int main() {
    int counts[] = {0, 0, 0, 0};
    int shots = 1000;
    for(int shot = 0; shot < shots; shot++) {
        int result = test_measure(shot);
        counts[result] += 1;
    }
    std::cout << "00: " << counts[0] << " " << 1.0f * counts[0] / shots <<std::endl;
    std::cout << "01: " << counts[1] << " " << 1.0f * counts[1] / shots <<std::endl;
    std::cout << "10: " << counts[2] << " " << 1.0f * counts[2] / shots <<std::endl;
    std::cout << "11: " << counts[3] << " " << 1.0f * counts[3] / shots <<std::endl;
}

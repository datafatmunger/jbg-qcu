
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

__global__ void setup_kernel(hiprandState *state, unsigned long long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void generate_random_numbers(hiprandState *state, float *randomNumbers)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Generate a random number using the curand_uniform function
    float randNum = hiprand_uniform(&state[idx]);
    
    // Store the random number in the output array
    randomNumbers[idx] = randNum;
}

// Function to swap two elements
void swap(float *xp, float *yp) {
    float temp = *xp;
    *xp = *yp;
    *yp = temp;
}

// Function to perform Bubble Sort
void bubbleSort(float arr[], int n) {
    int i, j;
    for (i = 0; i < n-1; i++) {
        for (j = 0; j < n-i-1; j++) {
            if (arr[j] > arr[j+1]) {
                swap(&arr[j], &arr[j+1]);
            }
        }
    }
}

// Function to count unique numbers in a sorted array
int countUnique(float arr[], int n) {
    if (n == 0) return 0;
    
    int count = 1; // there's at least one unique element
    for (int i = 1; i < n; i++) {
        if (arr[i] != arr[i-1]) {
            count++;
        }
    }
    return count;
}

int main()
{
    // Number of threads per block
    int threadsPerBlock = 256;
    // Number of blocks in the grid
    int blocksPerGrid = 256;
    // Total number of threads
    int numThreads = threadsPerBlock * blocksPerGrid;

    // Allocate memory for random number generator states
    hiprandState *devStates;
    hipMalloc((void **)&devStates, numThreads * sizeof(hiprandState));

    // Generate a random seed
    unsigned long long seed = time(NULL);

    // Setup the kernel with random states
    setup_kernel<<<blocksPerGrid, threadsPerBlock>>>(devStates, seed);
    
    // Allocate memory for the random numbers generated
    float *randomNumbers;
    hipMalloc((void **)&randomNumbers, numThreads * sizeof(float));

    // Generate random numbers using the initialized states
    generate_random_numbers<<<blocksPerGrid, threadsPerBlock>>>(devStates, randomNumbers);
    
    // Synchronize to ensure all kernel executions are complete
    hipDeviceSynchronize();

    // Copy the generated random numbers back to the host if needed
    float *hostRandomNumbers = (float *)malloc(numThreads * sizeof(float));
    hipMemcpy(hostRandomNumbers, randomNumbers, numThreads * sizeof(float), hipMemcpyDeviceToHost);

    //for(int i = 0; i < numThreads; i++) {
    //    std::cout << i << ": " << hostRandomNumbers[i] << std::endl;
    //}

    bubbleSort(hostRandomNumbers, numThreads);

    //for(int i = 0; i < numThreads; i++) {
    //    std::cout << i << ": " << hostRandomNumbers[i] << std::endl;
    //}

    int uniqueCount = countUnique(hostRandomNumbers, numThreads);
    std::cout << "Number of unique elements: " << uniqueCount << " in " << numThreads << std::endl;

    // Cleanup
    hipFree(devStates);
    hipFree(randomNumbers);

    free(hostRandomNumbers);

    return 0;
}

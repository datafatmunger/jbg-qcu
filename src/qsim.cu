#include "hip/hip_runtime.h"
#include "qsim.h"

#define BLOCK_SIZE 16

hipFloatComplex I::gateMatrix[H::matrixSize] = {
    make_hipFloatComplex(1, 0), make_hipFloatComplex(0, 0),
    make_hipFloatComplex(0, 0), make_hipFloatComplex(1, 0)
};

hipFloatComplex H::gateMatrix[H::matrixSize] = {
    make_hipFloatComplex(1, 0), make_hipFloatComplex(1, 0),
    make_hipFloatComplex(1, 0), make_hipFloatComplex(-1, 0)
};

hipFloatComplex CX::gateMatrix[CX::matrixSize] = {
    make_hipFloatComplex(1, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0),
    make_hipFloatComplex(0, 0), make_hipFloatComplex(1, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0),
    make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(1, 0),
    make_hipFloatComplex(0, 0), make_hipFloatComplex(0, 0), make_hipFloatComplex(1, 0), make_hipFloatComplex(0, 0),
    
};

// Error checking macro
#define cudaCheckError() {                                           \
    hipError_t e=hipGetLastError();                                \
    if(e!=hipSuccess) {                                             \
        std::cerr << "Cuda failure " << __FILE__ << ":" << __LINE__; \
        std::cerr << " '" << hipGetErrorString(e) << "'\n";         \
        exit(EXIT_FAILURE);                                          \
    }                                                                \
}

// CUDA kernel for complex matrix-vector multiplication
__global__ void matVecMulKernel(hipFloatComplex *A, hipFloatComplex *x, hipFloatComplex *y, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rows) {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int col = 0; col < cols; ++col) {
            sum = hipCaddf(sum, hipCmulf(A[row * cols + col], x[col]));
        }
        y[row] = sum;
    }
}

void matVecMul(hipFloatComplex *h_A, hipFloatComplex *h_x, hipFloatComplex *h_y, int rows, int cols) {
    hipFloatComplex *d_A, *d_x, *d_y;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, rows * cols * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_x, cols * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_y, rows * sizeof(hipFloatComplex));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, rows * cols * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, cols * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 dimBlock(1, BLOCK_SIZE);
    dim3 dimGrid(1, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the kernel
    matVecMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, rows, cols);

    // Copy the result back to the host
    hipMemcpy(h_y, d_y, rows * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}

__global__ void tensorProductKernel(hipFloatComplex* d_A, hipFloatComplex* d_B, hipFloatComplex* d_C, int aRows, int aCols, int bRows, int bCols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int totalRows = aRows * bRows;
    int totalCols = aCols * bCols;

    if (i < totalRows && j < totalCols) {
        int rowA = i / bRows;
        int colA = j / bCols;
        int rowB = i % bRows;
        int colB = j % bCols;
        d_C[i * totalCols + j] = hipCmulf(d_A[rowA * aCols + colA], d_B[rowB * bCols + colB]);
    }
}

void tensorProduct(hipFloatComplex* h_A, hipFloatComplex* h_B, hipFloatComplex* h_C, int aRows, int aCols, int bRows, int bCols) {
    int aSize = aRows * aCols * sizeof(hipFloatComplex);
    int bSize = bRows * bCols * sizeof(hipFloatComplex);
    int cSize = aRows * bRows * aCols * bCols * sizeof(hipFloatComplex);

    hipFloatComplex* d_A;
    hipFloatComplex* d_B;
    hipFloatComplex* d_C;

    hipMalloc((void**)&d_A, aSize);
    hipMalloc((void**)&d_B, bSize);
    hipMalloc((void**)&d_C, cSize);

    hipMemcpy(d_A, h_A, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((aCols * bCols + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (aRows * bRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    tensorProductKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows, bCols);

    hipMemcpy(h_C, d_C, cSize, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// Kernel to normalize the statevector
__global__ void normalize(hipFloatComplex* statevector, int len) {
    __shared__ float norm;
    if (threadIdx.x == 0) norm = 0.0f;
    __syncthreads();

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < len) {
        atomicAdd(&norm, hipCabsf(statevector[idx]) * hipCabsf(statevector[idx]));
    }
    __syncthreads();

    if (threadIdx.x == 0) norm = sqrtf(norm);
    __syncthreads();

    if (idx < len) {
        statevector[idx] = make_hipFloatComplex(hipCrealf(statevector[idx]) / norm, hipCimagf(statevector[idx]) / norm);
    }
}



// Kernel to compute the probabilities
__global__ void compute_probabilities(float* probabilities, hipFloatComplex* statevector, int len) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < len) {
        probabilities[idx] = hipCabsf(statevector[idx]) * hipCabsf(statevector[idx]);
    }
}

// Kernel to measure the statevector
__global__ void measure(int* result, float* probabilities, int len, unsigned long long seed) {
    hiprandState state;
    hiprand_init(seed, 0, 0, &state);
    float random_number = hiprand_uniform(&state);

    float cumulative_probability = 0.0f;
    for (int i = 0; i < len; ++i) {
        cumulative_probability += probabilities[i];
        if (random_number < cumulative_probability) {
            *result = i;
            break;
        }
    }
}

int measure(hipFloatComplex *h_statevector, int num_qubits, int shot) {
    int len = 1 << num_qubits;

    //hipFloatComplex h_statevector[] = {
    //    make_hipFloatComplex(0.2, 0.0), make_hipFloatComplex(0.2, 0.0),
    //    make_hipFloatComplex(0.6, 0.0), make_hipFloatComplex(0.2, 0.0)
    //};

    // // Calculate the norm of the statevector
    // float norm = 0.0f;
    // for (int i = 0; i < len; ++i) {
    //     float real_part = hipCrealf(h_statevector[i]);
    //     float imag_part = hipCimagf(h_statevector[i]);
    //     norm += real_part * real_part + imag_part * imag_part;
    // }
    // norm = sqrtf(norm);

    // // Normalize the statevector
    // for (int i = 0; i < len; ++i) {
    //     float real_part = hipCrealf(h_statevector[i]);
    //     float imag_part = hipCimagf(h_statevector[i]);
    //     h_statevector[i] = make_hipFloatComplex(real_part / norm, imag_part / norm);
    // }

    // // Verify normalization
    // float sum = 0.0f;
    // for (int i = 0; i < len; ++i) {
    //     float mag = hipCabsf(h_statevector[i]);
    //     sum += mag * mag;
    // }

    // Device memory allocations
    hipFloatComplex* d_statevector;
    float* d_probabilities;
    int* d_result;
    // Allocate memory for random number generator states
    //hiprandState *d_states;
    
    hipMalloc(&d_statevector, len * sizeof(hipFloatComplex));
    hipMalloc(&d_probabilities, len * sizeof(float));
    hipMalloc(&d_result, sizeof(int));
    //hipMalloc((void **)&d_states, len * sizeof(hiprandState));

    // Copy statevector to device
    hipMemcpy(d_statevector, h_statevector, len * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Kernel launches
    int blockSize = 256;
    int gridSize = (len + blockSize - 1) / blockSize;

    // Generate a random seed
    unsigned long long seed = time(NULL) + shot;

    normalize<<<gridSize, blockSize>>>(d_statevector, len);
    cudaCheckError();

    compute_probabilities<<<gridSize, blockSize>>>(d_probabilities, d_statevector, len);
    cudaCheckError();

    measure<<<1, 1>>>(d_result, d_probabilities, len, seed);
    cudaCheckError();

    // Copy result back to host
    int h_result;
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Convert result to bitstring
    //std::string bitstring;
    //for (int i = num_qubits - 1; i >= 0; --i) {
    //    bitstring += ((h_result >> i) & 1) ? '1' : '0';
    //}

    // Clean up
    hipFree(d_statevector);
    hipFree(d_probabilities);
    hipFree(d_result);

    //return binaryStringToInt(bitstring);
    return h_result;
}

// CUDA kernel to multiply a complex number against a matrix
__global__ void complexMatrixMultiply(const hipFloatComplex *matrix, const hipFloatComplex number, hipFloatComplex *result, int numRows, int numCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int index = row * numCols + col;
        
        // Perform complex multiplication
        float real = hipCrealf(matrix[index]) * hipCrealf(number) - hipCimagf(matrix[index]) * hipCimagf(number);
        float imag = hipCrealf(matrix[index]) * hipCimagf(number) + hipCimagf(matrix[index]) * hipCrealf(number);

        result[index] = make_hipFloatComplex(real, imag);
    }
}

int multiplication(hipFloatComplex *h_matrix, hipFloatComplex h_number, hipFloatComplex *h_result, int rows, int cols) {
    const int matrixSize = rows * cols;

    // Device (GPU) variables
    hipFloatComplex *d_matrix, *d_result;
    hipFloatComplex d_number = h_number;

    hipMalloc((void**)&d_matrix, matrixSize * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_result, matrixSize * sizeof(hipFloatComplex));

    // Transfer matrix and number to device
    hipMemcpy(d_matrix, h_matrix, matrixSize * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(32, 32);
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    complexMatrixMultiply<<<gridDim, blockDim>>>(d_matrix, d_number, d_result, rows, cols);

    // Copy result back to host
    hipMemcpy(h_result, d_result, matrixSize * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    hipFree(d_matrix);
    hipFree(d_result);

    return 0;
}


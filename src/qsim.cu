#include "hip/hip_runtime.h"
#include "qsim.h"

#define BLOCK_SIZE 16

// CUDA kernel for complex matrix-vector multiplication
__global__ void matVecMulKernel(hipFloatComplex *A, hipFloatComplex *x, hipFloatComplex *y, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rows) {
        hipFloatComplex sum = make_hipFloatComplex(0.0f, 0.0f);
        for (int col = 0; col < cols; ++col) {
            sum = hipCaddf(sum, hipCmulf(A[row * cols + col], x[col]));
        }
        y[row] = sum;
    }
}

void matVecMul(hipFloatComplex *h_A, hipFloatComplex *h_x, hipFloatComplex *h_y, int rows, int cols) {
    hipFloatComplex *d_A, *d_x, *d_y;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, rows * cols * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_x, cols * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_y, rows * sizeof(hipFloatComplex));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, rows * cols * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, cols * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 dimBlock(1, BLOCK_SIZE);
    dim3 dimGrid(1, (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the kernel
    matVecMulKernel<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, rows, cols);

    // Copy the result back to the host
    hipMemcpy(h_y, d_y, rows * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}

__global__ void tensorProductKernel(hipFloatComplex* d_A, hipFloatComplex* d_B, hipFloatComplex* d_C, int aRows, int aCols, int bRows, int bCols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int totalRows = aRows * bRows;
    int totalCols = aCols * bCols;

    if (i < totalRows && j < totalCols) {
        int rowA = i / bRows;
        int colA = j / bCols;
        int rowB = i % bRows;
        int colB = j % bCols;
        d_C[i * totalCols + j] = hipCmulf(d_A[rowA * aCols + colA], d_B[rowB * bCols + colB]);
    }
}

void tensorProduct(hipFloatComplex* h_A, hipFloatComplex* h_B, hipFloatComplex* h_C, int aRows, int aCols, int bRows, int bCols) {
    int aSize = aRows * aCols * sizeof(hipFloatComplex);
    int bSize = bRows * bCols * sizeof(hipFloatComplex);
    int cSize = aRows * bRows * aCols * bCols * sizeof(hipFloatComplex);

    hipFloatComplex* d_A;
    hipFloatComplex* d_B;
    hipFloatComplex* d_C;

    hipMalloc((void**)&d_A, aSize);
    hipMalloc((void**)&d_B, bSize);
    hipMalloc((void**)&d_C, cSize);

    hipMemcpy(d_A, h_A, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((aCols * bCols + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (aRows * bRows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    tensorProductKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, aRows, aCols, bRows, bCols);

    hipMemcpy(h_C, d_C, cSize, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
// add.cu

#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel function to add two numbers
__global__ void add(int* a, int* b, int* c) {
    *c = *a + *b;
}

int main() {
    // Host variables
    int a = 3;
    int b = 5;
    int c = 0;

    // Device variables
    int *d_a, *d_b, *d_c;

    // Allocate memory on the device
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // Copy host variables to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with one block and one thread
    add<<<1, 1>>>(d_a, d_b, d_c);

    // Copy the result back to the host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Result: " << c << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
